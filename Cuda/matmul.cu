
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
//Thread  block  size
#define BLOCK_SIZE 3
#define WA 10
//  Matrix A  width
#define HA 10
//  Matrix  A  height
#define WB 10
//  Matrix  B  width
#define HB WA
//  Matrix  B  height
#define WC WB
//  Matrix  C  width
#define HC HA
//  Matrix  C  height


//Allocates a  matrix  with  random  float  entries.
void randomInit(float * data ,int size)
{
	for(int i = 0; i < size; ++i)
		//data[i] = rand() / (float) RAND_MAX;
		data[i] = i;
}


//  CUDA  Kernel
__global__ void matrixMul(float* C,float* A,float* B,int hA, int wA,int wB)
{
	// 2D  Thread  ID
	int row = blockIdx.y*blockDim.y+threadIdx.y;
	int col = blockIdx.x*blockDim.x+threadIdx.x;
	//  value  stores  the  element  that  is  computed  by  the thread
	float value = 0;
	if(row < hA && col < wB)
	{
		for(int i = 0; i < wA; ++i)
		{
			float elementA = A[row * wA + i];
			float elementB = B[wA * i + col];
			value  +=  elementA * elementB;
		}
		//  Write  the  matrix  to  device  memory  each
		//  thread  writes  one  element
		C[row * wA + col] = value;
	}
}



//  Program  main
int main(int argc ,char** argv)
{
	//  set  seed  for  rand()
	srand(2006);
	//  1.  allocate  host  memory  for  matrices  A and B
	unsigned  int size_A = WA * HA;
	unsigned  int mem_size_A =sizeof(float) * size_A;
	float* h_A = (float*)  malloc(mem_size_A);
	unsigned  int size_B = WB * HB;
	unsigned  int mem_size_B =sizeof(float) * size_B;
	float * h_B = (float*)  malloc(mem_size_B);
	//  2.  initialize  host  memory
	randomInit(h_A, size_A);
	randomInit(h_B, size_B);
	//  3.  print  out A and B
	printf("\n\nMatrix A\n");
	for(int i = 0; i < size_A; i++)
	{
		printf("%6.0f ", h_A[i]);
		if(((i + 1) % WA) == 0)
			printf("\n");
	}
	printf("\n\nMatrix B\n");
	for(int i = 0; i < size_B; i++)
	{
		printf("%6.0f ", h_B[i]);
		if(((i + 1) % WB) == 0)
			printf("\n");
	}
	//  4.  allocate host  memory  for  the  result  C
	unsigned  int size_C = WC * HC;
	unsigned  int mem_size_C =sizeof(float) * size_C;
	float * h_C = (float *)  malloc(mem_size_C);

	//  8.  allocate  device  memory
	float* d_A;
	float* d_B;
	hipMalloc((void**) &d_A, mem_size_A);
	hipMalloc((void**) &d_B, mem_size_B);
	//9.  copy  host  memory  to  device
	hipMemcpy(d_A, h_A,mem_size_A ,hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B,mem_size_B ,hipMemcpyHostToDevice);
	//  10.  allocate  device  memory  for  the  result
	float* d_C;
	hipMalloc((void**) &d_C, mem_size_C);
	
	
	//  5.  perform  the  calculation
	//     setup  execution  parameters
	dim3  threads(BLOCK_SIZE , BLOCK_SIZE);
	dim3  grid((int)ceil((float)WC / threads.x), (int)ceil((float)HC / threads.y));
	//    execute  the  kernel
	matrixMul<<< grid , threads  >>>(d_C, d_A,d_B, HA, WA, WB);

	//  11.  copy  result  from  device  to  host
	hipMemcpy(h_C, d_C, mem_size_C ,hipMemcpyDeviceToHost);
	//  6.  print  out  the  results
	printf("\n\n Matrix C ( Results ) \n");
	for(int i = 0;i<size_C; i ++){
		printf("%6.0f ",h_C[i]);
		if(((i+ 1) % WC) == 0)
			printf("\n");
	}
	printf("\n");
	// 7.clean up memory
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	free(h_A);
	free(h_B);
	free(h_C);
}
