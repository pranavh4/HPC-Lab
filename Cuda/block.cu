
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define NUM_BLOCKS 32
#define BLOCK_WIDTH 1

__global__ void hello()
{
    printf("Hello world! I'm thread %d in block %d\n", threadIdx.x, blockIdx.x);
}


int main(int argc,char **argv)
{
    int num_blocks = NUM_BLOCKS, block_width = BLOCK_WIDTH;
    if(argc>1){
        num_blocks = atoi(argv[1]);
        block_width = atoi(argv[2]);
    }
    // launch the kernel
    hello<<<num_blocks, block_width>>>();

    // force the printf()s to flush
    hipDeviceSynchronize();
    
    return 0;
}
